#include "hip/hip_runtime.h"
char program[] = "CUDALucas v2.04 Alpha";
/* CUDALucas.c
   Shoichiro Yamada Oct. 2010 

   This is an adaptation of Richard Crandall lucdwt.c, John Sweeney MacLucasUNIX.c,
   and Guillermo Ballester Valor MacLucasFFTW.c code.
   Improvement From Prime95.
   
   It also contains mfaktc code by Oliver Weihe and Eric Christenson
   adapted for CUDALucas use. Such code is under the GPL, and is noted as such.
*/

/* Include Files */
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <time.h>
#include <signal.h>
#ifdef linux
#include <sys/types.h>
#include <sys/stat.h>
#else
#include <direct.h>
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "cuda_safecalls.h"
#include "parse.h"

/* In order to have the gettimeofday() function, you need these includes on Linux:
#include <sys/time.h>
#include <unistd.h>
On Windows, you need 
#include <winsock2.h> and a definition for
int gettimeofday (struct timeval *tv, struct timezone *) {}
Both platforms are taken care of in parse.h and parse.c. */

/************************ definitions ************************************/
/* global variables needed */
double *two_to_phi, *two_to_minusphi;
double *g_ttp, *g_ttmp;
char *g_numbits;
int *g_mask;
float *g_inv2, *g_inv3;
double *g_ttp2, *g_ttmp2, *g_ttp3, *g_ttmp3;
double high, low, highinv, lowinv;
double Gsmall, Gbig, Hsmall, Hbig;
hipfftHandle plan_fw, plan_bw, plan;
double *g_x, *g_save;
int j_save;
float *g_err;
int *g_carry;
int *ip, quitting, checkpoint_iter, b, c, fftlen, s_f, t_f, r_f, d_f, k_f;
int threads, polite, polite_f, bad_selftest=0;
char folder[132];
char input_filename[132], RESULTSFILE[132];
char INIFILE[132] = "CUDALucas.ini";
char s_residue[32];

/* http://www.kurims.kyoto-u.ac.jp/~ooura/fft.html
   base code from Takuya OOURA.  */
__global__ void
rftfsub_kernel (int n, double *a)
{
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  double wkr, wki, xr, xi, yr, yi, cc, d, aj, aj1, ak, ak1, *c;
  double new_aj, new_aj1, new_ak, new_ak1;
  const int m = n >> 1;
  const int nc = n >> 2;
  const int j = threadID << 1;
  const int j2 = threadID;
  c = &a[n];
  if (threadID)
    {
      int nminusj = n - j;

      wkr = 0.5 - c[nc - j2];
      wki = c[j2];
      aj = a[j];
      aj1 = a[1 + j];
      ak = a[nminusj];
      ak1 = a[1 + nminusj];
      xr = aj - ak;
      xi = aj1 + ak1;
      yr = wkr * xr - wki * xi;
      yi = wkr * xi + wki * xr;
      aj -= yr;
      aj1 -= yi;
      ak += yr;
      ak1 -= yi;

      new_aj1 = 2.0 * aj * aj1;
      new_aj = (aj - aj1) * (aj + aj1);

      new_ak1 = 2.0 * ak * ak1;
      new_ak = (ak - ak1) * (ak + ak1);

      xr = new_aj - new_ak;
      xi = new_aj1 + new_ak1;
      yr = wkr * xr + wki * xi;
      yi = wkr * xi - wki * xr;

      a[j] = new_aj - yr;
      a[1 + j] = yi - new_aj1;
      a[nminusj] = new_ak + yr;
      a[1 + nminusj] = yi - new_ak1;
    }
  else
    {
      xi = a[0] - a[1];
      a[0] += a[1];
      a[1] = xi;
      a[0] *= a[0];
      a[1] *= a[1];
      a[1] = 0.5 * (a[0] - a[1]);
      a[0] -= a[1];
      a[1] = -a[1];
      cc = a[0 + m];
      d = -a[1 + m];
      a[1 + m] = -2.0 * cc * d;
      a[0 + m] = (cc + d) * (cc - d);
      a[1 + m] = -a[1 + m];
    }
}

__global__ void
copy_kernel (double *save, double *x)
{
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  save[threadID] = x[threadID];
}

void
rdft (int n, int isgn, double *a, int *ip)
{
  void makect (int nc, int *ip, double *c);
  const int nc = n >> 2;
  int nw = ip[0];
  if (nw == 0)
    {
      makect (nc, ip, &a[n]);
      cutilSafeCall (hipMemcpy
		     (g_x, a, sizeof (double) * (n / 4 * 5),
		      hipMemcpyHostToDevice));
      if (t_f)
	copy_kernel <<< n / 128, 128 >>> (g_save, g_x);
    }
  cufftSafeCall (hipfftExecZ2Z
		 (plan, (hipfftDoubleComplex *) g_x,
		  (hipfftDoubleComplex *) g_x, HIPFFT_BACKWARD));
  rftfsub_kernel <<< n / 512, 128 >>> (n, g_x);
  cufftSafeCall (hipfftExecZ2Z
		 (plan, (hipfftDoubleComplex *) g_x,
		  (hipfftDoubleComplex *) g_x, HIPFFT_BACKWARD));
  return;
}

/* -------- initializing routines -------- */
void
makect (int nc, int *ip, double *c)
{
  int j;
  const int nch = nc >> 1;
  double delta;
  ip[0] = 1;
  ip[1] = nc;
  if (nc > 1)
    {
      delta = atan (1.0) / nch;
      c[0] = cos (delta * nch);
      c[nch] = 0.5 * c[0];
      for (j = 1; j < nch; j++)
	{
	  c[j] = 0.5 * cos (delta * j);
	  c[nc - j] = 0.5 * sin (delta * j);
	}
    }
}

/**************************************************************
 *
 *      FFT and other related Functions
 *
 **************************************************************/
/* rint is not ANSI compatible, so we need a definition for 
 * WIN32 and other platforms with rint.
 * Also we use that to write the trick to rint()
 */
# define RINT_x86(x) (floor(x+0.5))
# define RINT(x)  __rintd(x)
__device__ static double
__rintd (double x)
{
  double y;
asm ("cvt.rni.f64.f64 %0, %1;": "=d" (y):"d" (x));
  return (y);
}

#ifndef linux
long long int __double2ll (double);
#endif
__device__ static long long int
__double2ll (double x)
{
  long long int y;
asm ("cvt.rni.s64.f64 %0, %1;": "=l" (y):"d" (x));
  return (y);
}

/****************************************************************************
 *           Lucas Test - specific routines                                 *
 ***************************************************************************/
void
init_lucas (double *x, int q, int n)
{
  int j, qn, a, i, done;
  int size0, bj;
  double log2 = log (2.0);
  double ttp, ttmp;
  double *s_ttp, *s_ttmp;
  int *s_mask;
  float *s_inv;
  float *s_inv2;
  float *s_inv3;
  double *s_ttp2, *s_ttmp2;
  double *s_ttp3, *s_ttmp3;
  char *s_numbits;
  float *s_ttmpp;
  two_to_phi = (double *) malloc (sizeof (double) * (n / 2));
  two_to_minusphi = (double *) malloc (sizeof (double) * (n / 2));
  s_mask = (int *) malloc (sizeof (int) * 32);
  s_inv = (float *) malloc (sizeof (float) * (n));
  s_ttp = (double *) malloc (sizeof (double) * (n));
  s_ttmp = (double *) malloc (sizeof (double) * (n));
  s_ttmpp = (float *) malloc (sizeof (float) * (n));
  s_numbits = (char *) malloc (sizeof (char) * (n));
  s_inv2 = (float *) malloc (sizeof (float) * (n / threads));
  s_ttp2 = (double *) malloc (sizeof (double) * (n / threads));
  s_ttmp2 = (double *) malloc (sizeof (double) * (n / threads));
  s_inv3 = (float *) malloc (sizeof (float) * (n / threads));
  s_ttp3 = (double *) malloc (sizeof (double) * (n / threads));
  s_ttmp3 = (double *) malloc (sizeof (double) * (n / threads));
  cufftSafeCall (hipfftPlan1d (&plan, n / 2, HIPFFT_Z2Z, 1));
  cutilSafeCall (hipMalloc ((void **) &g_x, sizeof (double) * (n / 4 * 5)));
  if (t_f)
    cutilSafeCall (hipMalloc ((void **) &g_save, sizeof (double) * n));
  cutilSafeCall (hipMalloc ((void **) &g_err, sizeof (float)));
  cutilSafeCall (hipMalloc ((void **) &g_carry, sizeof (int) * n / threads));
  cutilSafeCall (hipMalloc ((void **) &g_mask, sizeof (int) * 32));
  cutilSafeCall (hipMalloc ((void **) &g_ttp, sizeof (double) * n));
  cutilSafeCall (hipMalloc ((void **) &g_ttmp, sizeof (double) * n));
  cutilSafeCall (hipMalloc ((void **) &g_numbits, sizeof (char) * n));
  cutilSafeCall (hipMalloc
		 ((void **) &g_inv2, sizeof (float) * n / threads));
  cutilSafeCall (hipMalloc
		 ((void **) &g_ttp2, sizeof (double) * n / threads));
  cutilSafeCall (hipMalloc
		 ((void **) &g_ttmp2, sizeof (double) * n / threads));
  cutilSafeCall (hipMalloc
		 ((void **) &g_inv3, sizeof (float) * n / threads));
  cutilSafeCall (hipMalloc
		 ((void **) &g_ttp3, sizeof (double) * n / threads));
  cutilSafeCall (hipMalloc
		 ((void **) &g_ttmp3, sizeof (double) * n / threads));
  cutilSafeCall (hipMemset (g_err, 0, sizeof (float)));
  low = floor ((exp (floor ((double) q / n) * log2)) + 0.5);
  high = low + low;
  lowinv = 1.0 / low;
  highinv = 1.0 / high;
  b = q % n;
  c = n - b;
  two_to_phi[0] = 1.0;
  two_to_minusphi[0] = 1.0 / (double) (n);
  qn = (b * 2) % n;
  for (i = 1, j = 2; j < n; j += 2, i++)
    {
      a = n - qn;
      two_to_phi[i] = exp (a * log2 / n);
      two_to_minusphi[i] = 1.0 / (two_to_phi[i] * n);
      qn += b * 2;
      qn %= n;
    }
  Hbig = exp (c * log2 / n);
  Gbig = 1 / Hbig;
  done = 0;
  j = 0;
  while (!done)
    {
      if (!((j * b) % n >= c || j == 0))
	{
	  a = n - ((j + 1) * b) % n;
	  i = n - (j * b) % n;
	  Hsmall = exp (a * log2 / n) / exp (i * log2 / n);
	  Gsmall = 1 / Hsmall;
	  done = 1;
	}
      j++;
    }
  bj = n;
  size0 = 1;
  bj = n - 1 * b;
  for (j = 0, i = 0; j < n; j = j + 2, i++)
    {
      ttmp = two_to_minusphi[i];
      ttp = two_to_phi[i];
      bj += b;
      bj = bj % n;
      size0 = (bj >= c);
      if (j == 0)
	size0 = 1;
      s_ttmp[j] = ttmp * 2.0;
      s_ttmpp[j] = (float) ttmp * n;
      if (size0)
	{
	  s_inv[j] = (float) highinv;
	  ttmp *= Gbig;
	  s_ttp[j] = ttp * high;
	  ttp *= Hbig;
	}
      else
	{
	  s_inv[j] = (float) lowinv;
	  ttmp *= Gsmall;
	  s_ttp[j] = ttp * low;
	  ttp *= Hsmall;
	}
      s_ttmpp[j] *= (float) s_ttp[j];
      bj += b;
      bj = bj % n;
      size0 = (bj >= c);
      if (j == (n - 2))
	size0 = 0;
      s_ttmp[j + 1] = ttmp * -2.0;
      s_ttmpp[j + 1] = (float) ttmp * n;
      if (size0)
	{
	  s_inv[j + 1] = (float) highinv;
	  s_ttp[j + 1] = ttp * high;
	}
      else
	{
	  s_inv[j + 1] = (float) lowinv;
	  s_ttp[j + 1] = ttp * low;
	}
      s_ttmpp[j + 1] *= (float) s_ttp[j + 1];
    }
  for (i = 0; i < n; i++)
    {
      s_ttmpp[i] = (float) ((long) (s_ttmpp[i] + 0.5));
      if (s_ttmpp[i] == s_ttmpp[0])
	s_numbits[i] = q / n + 1;
      else
	s_numbits[i] = q / n;
    }
  {
    for (i = 0; i < 32; i++)
      s_mask[i] = -1 << i;
    hipMemcpy (g_mask, s_mask, sizeof (int) * 32, hipMemcpyHostToDevice);
    hipMemcpy (g_ttmp, s_ttmp, sizeof (double) * n, hipMemcpyHostToDevice);
    hipMemcpy (g_numbits, s_numbits, sizeof (char) * n,
		hipMemcpyHostToDevice);
  }
  for (i = 0, j = 0; i < n; i++)
    {
      if ((i % threads) == 0)
	{
	  s_inv2[j] = s_inv[i];
	  s_ttp2[j] = s_ttp[i];
	  s_ttmp2[j] = s_ttmp[i] * 0.5 * n;
	  s_inv3[j] = s_inv[i + 1];
	  s_ttp3[j] = s_ttp[i + 1];
	  s_ttmp3[j] = s_ttmp[i + 1] * (-0.5) * n;
	  j++;
	}
    }
  for (i = 0, j = 0; i < n; i++)
    s_ttp[i] *= s_inv[i];
  hipMemcpy (g_ttp, s_ttp, sizeof (double) * n, hipMemcpyHostToDevice);
  hipMemcpy (g_inv2, s_inv2, sizeof (float) * n / threads,
	      hipMemcpyHostToDevice);
  hipMemcpy (g_ttp2, s_ttp2, sizeof (double) * n / threads,
	      hipMemcpyHostToDevice);
  hipMemcpy (g_ttmp2, s_ttmp2, sizeof (double) * n / threads,
	      hipMemcpyHostToDevice);
  hipMemcpy (g_inv3, s_inv3, sizeof (float) * n / threads,
	      hipMemcpyHostToDevice);
  hipMemcpy (g_ttp3, s_ttp3, sizeof (double) * n / threads,
	      hipMemcpyHostToDevice);
  hipMemcpy (g_ttmp3, s_ttmp3, sizeof (double) * n / threads,
	      hipMemcpyHostToDevice);

  free ((char *) s_inv);
  free ((char *) s_ttp);
  free ((char *) s_mask);
  free ((char *) s_ttmp);
  free ((char *) s_ttmpp);
  free ((char *) s_inv2);
  free ((char *) s_ttp2);
  free ((char *) s_ttmp2);
  free ((char *) s_inv3);
  free ((char *) s_ttp3);
  free ((char *) s_ttmp3);
  free ((char *) s_numbits);

  ip = (int *) malloc (((size_t) (2 + sqrt ((float) n / 2)) * sizeof (int)));
  ip[0] = 0;
}

void
close_lucas (double *x)
{
  free ((char *) x);
  free ((char *) two_to_phi);
  free ((char *) two_to_minusphi);
  free ((char *) ip);
  cutilSafeCall (hipFree ((char *) g_x));
  if (t_f)
    cutilSafeCall (hipFree ((char *) g_save));
  cutilSafeCall (hipFree ((char *) g_err));
  cutilSafeCall (hipFree ((char *) g_carry));
  cutilSafeCall (hipFree ((char *) g_mask));
  cutilSafeCall (hipFree ((char *) g_ttp));
  cutilSafeCall (hipFree ((char *) g_ttmp));
  cutilSafeCall (hipFree ((char *) g_numbits));
  cutilSafeCall (hipFree ((char *) g_inv2));
  cutilSafeCall (hipFree ((char *) g_ttp2));
  cutilSafeCall (hipFree ((char *) g_ttmp2));
  cutilSafeCall (hipFree ((char *) g_inv3));
  cutilSafeCall (hipFree ((char *) g_ttp3));
  cutilSafeCall (hipFree ((char *) g_ttmp3));
  cufftSafeCall (hipfftDestroy (plan));
}

template < int g_err_flag > __global__ void
normalize_kernel (double *g_x, int threads,
		  volatile float *g_err, int *g_carry, int *g_mask,
		  double *g_ttp, double *g_ttmp, char *g_numbits,
		  float maxerr)
{
  long long int bigint;
  int val, numbits, mask, shifted_carry;
  __shared__ int carry[1024 + 1];
  // read the matrix tile into shared memory
  unsigned int index = blockIdx.x * threads + threadIdx.x;
  if (g_err_flag)
    {
      double tval, trint;
      float ferr;
//0
      tval = g_x[index] * g_ttmp[index];
      trint = RINT (tval);
      ferr = tval - trint;
      ferr = fabs (ferr);

      bigint = trint;

      if (ferr > maxerr) 
	   atomicMax((int*)g_err, __float_as_int(ferr));
    }
  else
    {
//0
      bigint = __double2ll (g_x[index] * g_ttmp[index]);
    }

  numbits = g_numbits[index];
  carry[threadIdx.x + 1] = (int) (bigint >> numbits);
  mask = g_mask[numbits];
  val = ((int) bigint) & ~mask;

//1    
  __syncthreads ();
  if (threadIdx.x)
    val += carry[threadIdx.x];
  shifted_carry = val - g_mask[numbits - 1];
  carry[threadIdx.x] = shifted_carry >> numbits;
  val = val - (shifted_carry & mask);

  if (threadIdx.x == (threads - 1))
    g_carry[blockIdx.x] = carry[threadIdx.x + 1] + carry[threadIdx.x];

//2
  __syncthreads ();
  if (threadIdx.x)
    val += carry[threadIdx.x - 1];
  g_x[index] = (double) val *g_ttp[index];

}

__global__ void
normalize2_kernel (double *g_x, int threads, int *g_carry,
		   int g_N, float *g_inv2, double *g_ttp2, double *g_ttmp2,
		   float *g_inv3, double *g_ttp3, double *g_ttmp3)
{
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = threads * threadID;
  double temp0, tempErr;
  double temp1, tempErr2;
  int carry;
  if (j < g_N)
    {
      if (threadID)
	carry = g_carry[threadID - 1];
      else
	carry = g_carry[g_N / threads - 1] - 2;	// The -2 is part of the LL test
      temp0 = g_x[j];
      temp1 = g_x[j + 1];
      tempErr = temp0 * g_ttmp2[threadID];
      tempErr2 = temp1 * g_ttmp3[threadID];
      temp0 = tempErr + carry;
      temp0 *= g_inv2[threadID];
      carry = RINT (temp0);
      temp1 = tempErr2 + carry;
      temp1 *= g_inv3[threadID];
      g_x[j] = (temp0 - carry) * g_ttp2[threadID];
      g_x[j + 1] = temp1 * g_ttp3[threadID];
    }
}

double
last_normalize (double *x, int N, int err_flag)
{
  int i, j, k, bj, size0;
  double hi = high, hiinv = highinv, lo = low, loinv = lowinv, temp0, tempErr;
  double err = 0.0, terr = 0.0, ttmpSmall = Gsmall, ttmpBig =
    Gbig, ttmp, carry;
  carry = -2.0;			/* this is the -2 of the LL x*x - 2 */
  bj = N;
  size0 = 1;
  for (j = 0, i = 0; j < N; j += 2, i++)
    {
      ttmp = two_to_minusphi[i];
      temp0 = x[j];
      temp0 *= 2.0;
      tempErr = RINT_x86 (temp0 * ttmp);
      if (err_flag)
	{
	  terr = fabs (temp0 * ttmp - tempErr);
	  if (terr > err)
	    err = terr;
	}
      temp0 = tempErr + carry;
      if (size0)
	{
	  temp0 *= hiinv;
	  carry = RINT_x86 (temp0);
	  bj += b;
	  ttmp *= ttmpBig;
	  if (bj >= N)
	    bj -= N;
	  x[j] = (temp0 - carry) * hi;
	  size0 = (bj >= c);
	}
      else
	{
	  temp0 *= loinv;
	  carry = RINT_x86 (temp0);
	  bj += b;
	  ttmp *= ttmpSmall;
	  if (bj >= N)
	    bj -= N;
	  x[j] = (temp0 - carry) * lo;
	  size0 = (bj >= c);
	}
      temp0 = x[j + 1];
      temp0 *= -2.0;

      if (j == N - 2)
	size0 = 0;
      tempErr = RINT_x86 (temp0 * ttmp);
      if (err_flag)
	{
	  terr = fabs (temp0 * ttmp - tempErr);
	  if (terr > err)
	    err = terr;
	}
      temp0 = tempErr + carry;
      if (size0)
	{
	  temp0 *= hiinv;
	  carry = RINT_x86 (temp0);
	  bj += b;
	  ttmp *= ttmpBig;
	  if (bj >= N)
	    bj -= N;
	  x[j + 1] = (temp0 - carry) * hi;
	  size0 = (bj >= c);
	}
      else
	{
	  temp0 *= loinv;
	  carry = RINT_x86 (temp0);
	  bj += b;
	  ttmp *= ttmpSmall;
	  if (bj >= N)
	    bj -= N;
	  x[j + 1] = (temp0 - carry) * lo;
	  size0 = (bj >= c);
	}
    }
  bj = N;
  k = 0;
  while (carry != 0)
    {
      size0 = (bj >= c);
      bj += b;
      temp0 = (x[k] + carry);
      if (bj >= N)
	bj -= N;
      if (size0)
	{
	  temp0 *= hiinv;
	  carry = RINT_x86 (temp0);
	  x[k] = (temp0 - carry) * hi;
	}
      else
	{
	  temp0 *= loinv;
	  carry = RINT_x86 (temp0);
	  x[k] = (temp0 - carry) * lo;
	}
      k++;
    }
  return (err);
}

double
lucas_square (double *x, int N, int iter, int last, float maxerr,
	      int error_flag)
{
  double terr;
  rdft (N, 1, x, ip);
  if (iter == last)
    {
      cutilSafeCall (hipMemcpy
		     (x, g_x, sizeof (double) * N, hipMemcpyDeviceToHost));
      terr = last_normalize (x, N, error_flag);
    }
  else
    {

      if ((iter % checkpoint_iter) == 0)
	{
	  cutilSafeCall (hipMemcpy
			 (x, g_x, sizeof (double) * N,
			  hipMemcpyDeviceToHost));
	  terr = last_normalize (x, N, error_flag);
	}

      if (error_flag || t_f)
	{
	  normalize_kernel < 1 > <<<N / threads, threads >>> (g_x, threads,
							      g_err, g_carry,
							      g_mask, g_ttp,
							      g_ttmp,
							      g_numbits,
							      maxerr);
	}
      else
	{
	  normalize_kernel < 0 > <<<N / threads, threads >>> (g_x, threads,
							      g_err, g_carry,
							      g_mask, g_ttp,
							      g_ttmp,
							      g_numbits,
							      maxerr);
	}
      normalize2_kernel <<< ((N + threads - 1) / threads + 127) / 128,
	128 >>> (g_x, threads, g_carry, N, g_inv2, g_ttp2, g_ttmp2, g_inv3,
		 g_ttp3, g_ttmp3);
      {
	float l_err;
	if (polite_f && (iter % polite) == 0)
	  cutilSafeCall (hipMemcpy
			 (&l_err, g_err, sizeof (float),
			  hipMemcpyDeviceToHost));
      }
      terr = 0.0;
      if (error_flag)
	{
	  float c_err;
	  cutilSafeCall (hipMemcpy
			 (&c_err, g_err, sizeof (float),
			  hipMemcpyDeviceToHost));
	  terr = c_err;
	}
    }
  return (terr);
}

int
choose_fft_length (int input_length)
{ 
  #ifdef TEST
  printf("FFT selector called on %d\n", input_length);
  #endif
  int np[13] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 12, 14, 15 };
  int output_length = 1;
  int i, tmp;
  do
    {
      #ifdef TEST
      printf("Output_length is now %d\n", output_length);
      #endif
      for (i = 0; i < 13; i++)
      {
        tmp = output_length * np[i];
        #ifdef TEST
        printf("Output_length * np[%d] is %d\n", i, tmp);
        #endif
	   if ( tmp >= input_length) {
	   #ifdef TEST
	   printf("%d is greater than input %d, returning %d, which is %dK + %d\n", tmp, input_length, tmp, tmp/1024, tmp%1024);
	   #endif
	     return (int) tmp;
	   }
	 }
    }
  while (output_length *= 2);
  return 0;
}

//From apsen
void
print_time_from_seconds (int sec)
{
  if (sec > 3600)
    {
      printf ("%d", sec / 3600);
      sec %= 3600;
      printf (":%02d", sec / 60);
    }
  else
    printf ("%d", sec / 60);
  sec %= 60;
  printf (":%02d", sec);
}

void
init_device (int device_number)
{
  int device_count = 0;
  struct hipDeviceProp_t properties;
  hipGetDeviceCount (&device_count);
  if (device_number >= device_count)
    {
      printf ("device_number >=  device_count ... exiting\n\n");
      exit (2);
    }
  if (d_f)
    {
      hipDeviceProp_t dev;
      hipGetDeviceProperties (&dev, device_number);
      printf ("------- DEVICE %d -------\n", device_number);
      printf ("name                %s\n", dev.name);
      printf ("totalGlobalMem      %d\n", (int) dev.totalGlobalMem);
      printf ("sharedMemPerBlock   %d\n", (int) dev.sharedMemPerBlock);
      printf ("regsPerBlock        %d\n", (int) dev.regsPerBlock);
      printf ("warpSize            %d\n", (int) dev.warpSize);
      printf ("memPitch            %d\n", (int) dev.memPitch);
      printf ("maxThreadsPerBlock  %d\n", (int) dev.maxThreadsPerBlock);
      printf
	("maxThreadsDim[3]    %d,%d,%d\n",
	 dev.maxThreadsDim[0], dev.maxThreadsDim[1], dev.maxThreadsDim[2]);
      printf
	("maxGridSize[3]      %d,%d,%d\n", dev.maxGridSize[0],
	 dev.maxGridSize[1], dev.maxGridSize[2]);
      printf ("totalConstMem       %d\n", (int) dev.totalConstMem);
      printf ("Compatibility       %d.%d\n", dev.major, dev.minor);
      printf ("clockRate (MHz)     %d\n", dev.clockRate/1000);
      printf ("textureAlignment    %d\n", (int) dev.textureAlignment);
      printf ("deviceOverlap       %d\n", dev.deviceOverlap);
      printf ("multiProcessorCount %d\n\n", dev.multiProcessorCount);
    }
  hipSetDeviceFlags (hipDeviceScheduleBlockingSync);
  hipSetDevice (device_number);
// From Iain
  hipGetDeviceProperties (&properties, device_number);

  if (properties.major == 1 && properties.minor < 3)
    {
      printf
	("A GPU with compute capability >= 1.3 is required for double precision arithmetic\n\n");
      exit (2);
    }
}

int
is_big2 (int j, int bigx, int smallx, int n)
{
  return ((((bigx * j) % n) >= smallx) || j == 0);
}

void
balancedtostdrep (double *x, int n, int b, int c, double hi, double lo,
		  int mask, int shift)
{
  int sudden_death = 0, j = 0, NminusOne = n - 1, k, k1;
  while (1)
    {
      k = j + ((j & mask) >> shift);
      if (x[k] < 0.0)
	{
	  k1 = (j + 1) % n;
	  k1 += (k1 & mask) >> shift;
	  --x[k1];
	  if (j == 0 || (j != NminusOne && is_big2 (j, b, c, n)))
	    x[k] += hi;
	  else
	    x[k] += lo;
	}
      else if (sudden_death)
	break;
      if (++j == n)
	{
	  sudden_death = 1;
	  j = 0;
	}
    }
}

int
is_zero (double *x, int n, int mask, int shift)
{
  int j, offset;
  for (j = 0; j < n; ++j)
    {
      offset = j + ((j & mask) >> shift);
      if (rint (x[offset]))
	return (0);
    }
  return (1);
}

int
printbits (double *x,
	   int q,
	   int N,
	   int b, int c, double high, double low, int totalbits,
	   int flag, char *expectedResidue)
{
  char *bits = (char *) malloc ((int) totalbits);
  char residue[32];
  char temp[32];
  int j, k, i, word;
  FILE *fp=NULL;
  if (flag)
    {
      fp = fopen (RESULTSFILE, "a");
      if (fp == NULL)
	{
	  fprintf (stderr, "Cannot write results to %s\n\n", RESULTSFILE);
	  exit (1);
	}
    }
  if (is_zero (x, N, 0, 0))
    {
      printf ("M( %d )P, n = %d, %s", q, N, program);
      if (flag)
	{
	  fprintf (fp, "M( %d )P, n = %d, %s", q, N, program);
	  fprintf (fp, "\n");
	  fclose (fp);
	}
    }
  else
    {
      double *x_tmp;
      x_tmp = (double *) malloc (sizeof (double) * N);
      for (i = 0; i < N; i++)
	x_tmp[i] = x[i];
      balancedtostdrep (x_tmp, N, b, c, high, low, 0, 0);
      printf ("M( %d )C, 0x", q);
      if (flag)
	fprintf (fp, "M( %d )C, 0x", q);
      j = 0;
      i = 0;
      do
	{
	  k = (int) (ceil ((double) q * (j + 1) / N) -
		     ceil ((double) q * j / N));
	  if (k > totalbits)
	    k = totalbits;
	  totalbits -= k;
	  word = (int) x_tmp[j++];
	  while (k--)
	    {
	      bits[i++] = (char) ('0' + (word & 1));
	      word >>= 1;
	    }
	}
      while (totalbits);
      residue[0] = 0;
      while (i)
	{
	  k = 0;
	  for (j = 0; j < 4; j++)
	    {
	      i--;
	      k <<= 1;
	      if (bits[i] == '1')
		k++;
	    }
	  if (k > 9)
	    {
	      sprintf (temp, "%s", residue);
	      sprintf (residue, "%s%c", temp, (char) ('a' + k - 10));
	    }
	  else
	    {
	      sprintf (temp, "%s", residue);
	      sprintf (residue, "%s%c", temp, (char) ('0' + k));
	    }
	}
      free (x_tmp);
      printf ("%s", residue);
      printf (", n = %d, %s", N, program);
      if (flag)
	{
	  fprintf (fp, "%s", residue);
	  fprintf (fp, ", n = %d, %s", N, program);
	  fprintf (fp, "\n");
	  fclose (fp);
	}
      if (expectedResidue && strcmp (residue, expectedResidue))
      {
	   bad_selftest++;
	   return 1;
      }
      else if(expectedResidue) 
	{
	   return 0;
	}
    } /* end else res not 0 */
  sprintf (s_residue, "%s", residue);
  free (bits);
  return 0;
}

void
rm_checkpoint (int q)
{
  char chkpnt_cfn[32];
  char chkpnt_tfn[32];
  sprintf (chkpnt_cfn, "c" "%d", q);
  sprintf (chkpnt_tfn, "t" "%d", q);
  (void) unlink (chkpnt_cfn);
  (void) unlink (chkpnt_tfn);
}

double *
read_checkpoint (int q, int *n, int *j)
{
  FILE *fPtr;
  int q_r, n_r, j_r;
  double *x;
  char chkpnt_cfn[32];
  char chkpnt_tfn[32];
  sprintf (chkpnt_cfn, "c" "%d", q);
  sprintf (chkpnt_tfn, "t" "%d", q);
  fPtr = fopen (chkpnt_cfn, "rb");
  if (!fPtr)
    {
      fPtr = fopen (chkpnt_tfn, "rb");
      if (!fPtr)
	return NULL;
    }
  // check parameters
  if (fread (&q_r, 1, sizeof (q_r), fPtr) != sizeof (q_r)
      || fread (&n_r, 1, sizeof (n_r), fPtr)
      != sizeof (n_r) || fread (&j_r, 1, sizeof (j_r), fPtr) != sizeof (j_r))
    {
      fprintf (stderr,
	       "\nThe checkpoint doesn't match current test.  Current test will be restarted\n");
      fclose (fPtr);
      return NULL;
    }
  if (q != q_r)
    {
      fprintf
	(stderr,
	 "\nThe checkpoint doesn't match current test.  Current test will be restarted\n");
      fclose (fPtr);
      return NULL;
    }
  // check for successful read of z, delayed until here since zSize can vary
  x = (double *) malloc (sizeof (double) * (n_r + n_r));
  if (fread (x, 1, sizeof (double) * (n_r), fPtr) !=
      (sizeof (double) * (n_r)))
    {
      fprintf (stderr,
	       "\nThe checkpoint doesn't match current test.  Current test will be restarted\n");
      fclose (fPtr);
      free (x);
      return NULL;
    }
  // have good stuff, do checkpoint
  *n = n_r;
  *j = j_r;
  fclose (fPtr);
  return x;
}

void
write_checkpoint (double *x, int q, int n, int j)
{
  FILE *fPtr;
  char chkpnt_cfn[32];
  char chkpnt_tfn[32];
  sprintf (chkpnt_cfn, "c" "%d", q);
  sprintf (chkpnt_tfn, "t" "%d", q);
  (void) unlink (chkpnt_tfn);
  (void) rename (chkpnt_cfn, chkpnt_tfn);
  fPtr = fopen (chkpnt_cfn, "wb");
  if (!fPtr)
    return;
  fwrite (&q, 1, sizeof (q), fPtr);
  fwrite (&n, 1, sizeof (n), fPtr);
  fwrite (&j, 1, sizeof (j), fPtr);
  fwrite (x, 1, sizeof (double) * n, fPtr);
  fclose (fPtr);
  if (s_f > 0)			// save all checkpoint files
    {
      char chkpnt_sfn[64];
#ifdef linux
      sprintf (chkpnt_sfn, "%s/s" "%d.%d.%s", folder, q, j, s_residue);
#else
      sprintf (chkpnt_sfn, "%s\\s" "%d.%d.%s.txt", folder, q, j, s_residue);
#endif
      fPtr = fopen (chkpnt_sfn, "wb");
      if (!fPtr)
	return;
      fwrite (&q, 1, sizeof (q), fPtr);
      fwrite (&n, 1, sizeof (n), fPtr);
      fwrite (&j, 1, sizeof (j), fPtr);
      fwrite (x, 1, sizeof (double) * n, fPtr);
      fclose (fPtr);
    }
}

void
cufftbench (int cufftbench_s, int cufftbench_e, int cufftbench_d)
{
  hipEvent_t start, stop;
  double *x;
  float outerTime;
  int i, j;
  printf ("CUFFT bench start = %d end = %d distance = %d\n", cufftbench_s,
	  cufftbench_e, cufftbench_d);

  cutilSafeCall (hipMalloc ((void **) &g_x, sizeof (double) * cufftbench_e));
  x = ((double *) malloc (sizeof (double) * cufftbench_e + 1));
  for (i = 0; i <= cufftbench_e; i++)
    x[i] = 0;
  cutilSafeCall (hipMemcpy
		 (g_x, x, sizeof (double) * cufftbench_e,
		  hipMemcpyHostToDevice));
  cutilSafeCall (hipEventCreate (&start));
  cutilSafeCall (hipEventCreate (&stop));
  for (j = cufftbench_s; j <= cufftbench_e; j += cufftbench_d)
    {
      cufftSafeCall (hipfftPlan1d (&plan, j / 2, HIPFFT_Z2Z, 1));
      cufftSafeCall (hipfftExecZ2Z
		     (plan, (hipfftDoubleComplex *) g_x,
		      (hipfftDoubleComplex *) g_x, HIPFFT_BACKWARD));
      cutilSafeCall (hipEventRecord (start, 0));
      for (i = 0; i < 100; i++)
	cufftSafeCall (hipfftExecZ2Z
		       (plan, (hipfftDoubleComplex *) g_x,
			(hipfftDoubleComplex *) g_x, HIPFFT_BACKWARD));
      cutilSafeCall (hipEventRecord (stop, 0));
      cutilSafeCall (hipEventSynchronize (stop));
      cutilSafeCall (hipEventElapsedTime (&outerTime, start, stop));
      printf ("HIPFFT_Z2Z size= %d time= %f msec\n", j, outerTime / 100);
      cufftSafeCall (hipfftDestroy (plan));
    }
  cutilSafeCall (hipFree ((char *) g_x));
  cutilSafeCall (hipEventDestroy (start));
  cutilSafeCall (hipEventDestroy (stop));
  free ((char *) x);
}

void
SetQuitting (int sig)
{
  quitting = 1;
 sig==SIGTERM ? fprintf(stderr, "\nSIGTERM") : (sig==SIGINT ? fprintf(stderr, "\nSIGINT") : fprintf(stderr, "\nUnknown signal")) ;
 fprintf(stderr, " caught. Writing checkpoint.\n\n");
}

#ifdef linux
#include <termios.h>
#include <unistd.h>
#include <fcntl.h>
int
_kbhit (void)
{
  struct termios oldt, newt;
  int ch;
  int oldf;

  tcgetattr (STDIN_FILENO, &oldt);
  newt = oldt;
  newt.c_lflag &= ~(ICANON | ECHO);
  tcsetattr (STDIN_FILENO, TCSANOW, &newt);
  oldf = fcntl (STDIN_FILENO, F_GETFL, 0);
  fcntl (STDIN_FILENO, F_SETFL, oldf | O_NONBLOCK);

  ch = getchar ();

  tcsetattr (STDIN_FILENO, TCSANOW, &oldt);
  fcntl (STDIN_FILENO, F_SETFL, oldf);

  if (ch != EOF)
    {
      ungetc (ch, stdin);
      return 1;
    }

  return 0;
}
#else
#include <conio.h>
#endif
/**************************************************************
 *
 *      Main Function
 *
 **************************************************************/
int
check (int q, char *expectedResidue)
{
  int n = q/20, j = 1L, last = 2L, error_flag;
  size_t k;
  double terr, *x = NULL, maxerr;
  int restarting = 0;
  timeval time0, time1;
  if (!expectedResidue)
    {
      // We log to file in most cases anyway.
      signal (SIGTERM, SetQuitting);
      signal (SIGINT, SetQuitting);
    }
  do
    {				/* while (restarting) */
      maxerr = 0.0;
      if (fftlen)
	   n = fftlen;
	 else
	   #ifdef TEST
        print("Exp = %d, Exp/20 = %d\n", q, q/20);
        #endif
	   n = choose_fft_length( n );
      if ((n / threads) > 65535)
	{
	  fprintf (stderr, "over specifications Grid = %d\n", (int) n / threads);
	  fprintf (stderr, "try increasing threads or decreasing FFT length\n\n");
	  exit (2);
	}
      if (!expectedResidue && !restarting
	  && (x = read_checkpoint (q, &n, &j)) != NULL)
	printf
	  ("Continuing work from a partial result of M%d fft length = %d iteration = %d\n",
	   q, n, j);
      else
	{
	  printf ("Starting M%d fft length = %d\n", q, n);
	  x = (double *) malloc (sizeof (double) * (n + n));
	  for (k = 1; k < (unsigned int)n; k++)
	    x[k] = 0.0;
	  x[0] = 4.0;
	  j = 1;
	  if (t_f)
	    j_save = 0;
	}
      fflush (stdout);
      restarting = 0;
      init_lucas (x, q, n);
      gettimeofday (&time0, NULL);
      last = q - 2;		/* the last iteration done in the primary loop */

      for (; !restarting && j <= last; j++)
	{
	  if ((j % 100) == 1 || j < 1000)
	    error_flag = 1;
	  else
	    error_flag = 0;

	  terr = lucas_square (x, n, j, last, (float) maxerr, error_flag);

	  if (error_flag)
	    {
	      if (terr > maxerr)
		maxerr = terr;
	      if (j < 1000)
		{
		  if (terr >= 0.25)
		    {
		      if (!fftlen)
			{	/* n is not big enough; increase it and start over */
			  printf
			    ("iteration = %d < 1000 && err = %g >= 0.25, increasing n from %d\n",
			     j, (double) terr, (int) n);
			  n++;
			  restarting = 1;
			}
		    }
		}
	      else		// error_flag && j >= 1000
		{
		  if (terr >= 0.35)
		    {
		      if (t_f)
			{
			  printf
			    ("iteration = %d >= 1000 && err = %g >= 0.35, fft length = %d, writing checkpoint file (because -t is enabled) and exiting.\n\n",
			     j, (double) terr, (int) n);
			  cutilSafeCall (hipMemcpy
					 (x, g_save, sizeof (double) * n,
					  hipMemcpyDeviceToHost));
			  write_checkpoint (x, q, n, j_save + 1);
			  exit (2);
			}
		      else
			{
			  printf
			    ("iteration = %d >= 1000 && err = %g >= 0.35, fft length = %d, not writing checkpoint file (because -t is disabled) and exiting.\n\n",
			     j, (double) terr, (int) n);
			  exit (2);
			}
		    }
		  else		// error_flag && j >= 1000 && terr < 0.35
		    {
		      if (t_f)
			{
			  copy_kernel <<< n / 128, 128 >>> (g_save, g_x);
			  j_save = j;
			}
		    }
		}
	    }
	  if ((j % checkpoint_iter) == 0)
	    {
	      gettimeofday (&time1, NULL);
	      printf ("Iteration %d ", j);
	      int ret = printbits (x, q, n, b, c, high, low, 64, 0, expectedResidue);
	      long diff = time1.tv_sec - time0.tv_sec;
	      long diff1 = 1000000 * diff + time1.tv_usec - time0.tv_usec;
	      printf (" err = %4.4f (", maxerr);
	      print_time_from_seconds (diff);
	      printf (" real, %4.4f ms/iter, ETA ",
		      diff1 / 1000.0 / checkpoint_iter);
	      diff = (long) ((last - j) / checkpoint_iter * (diff1 / 1e6));
	      print_time_from_seconds (diff);
	      printf (")\n");
	      fflush (stdout);
	      gettimeofday (&time0, NULL);
	      if (expectedResidue) 
	      {
		j = last + 1;
		if (ret)
		  printf
		  ("\nExpected residue [%s] does not match actual residue [%s]\n",
	          expectedResidue, s_residue);
	        else printf("This residue is correct.\n");
	      }
	    }

	  if (((j % checkpoint_iter) == 0 || quitting == 1)
	      && !expectedResidue)
	    {
	      cutilSafeCall (hipMemcpy
			     (x, g_x, sizeof (double) * n,
			      hipMemcpyDeviceToHost));
	      write_checkpoint (x, q, n, j + 1);
	      if (quitting == 1)
		j = last + 1;
	    }

	  if (k_f && !quitting && !expectedResidue && (!(j & 15))
	      && _kbhit ())
	    {
	      int c = getchar ();
	      if (c == 'p')
		if (polite_f)
		  {
		    polite_f = 0;
		    printf ("   -polite 0\n");
		  }
		else
		  {
		    polite_f = 1;
		    printf ("   -polite %d\n", polite);
		  }
	      if (c == 't')
		{
		  t_f = 0;
		  printf ("   disabling -t\n");
		}
	      if (c == 's')
		if (s_f == 1)
		  {
		    s_f = 2;
		    printf ("   disabling -s\n");
		  }
		else if (s_f == 2)
		  {
		    s_f = 1;
		    printf ("   enabling -s\n");
		  }
	      fflush (stdin);
	    }
	} /* end main LL for-loop */
      if (!restarting && !expectedResidue && !quitting)
	{
	  printbits (x, q, n, b, c, high, low, 64, 1, 0);
	  printf ("\n");
	  fflush (stdout);
	  rm_checkpoint (q);
	}
      close_lucas (x);
    }
  while (restarting);
  return (0);
}

void parse_args(int* argc, char* *argv[], int* q, int* device_numer, 
		int* cufftbench_s, int* cufftbench_e, int* cufftbench_d);
		/* The rest of the opts are global */
int main (int argc, char *argv[])
{ 
  printf("\n");
  quitting = 0;

/*! Old default settings; kept here just in case.
  sprintf (input_filename, "");
  checkpoint_/iter = 10000;
  threads = 256;
  fftlen = 0;
  quitting = 0;
  s_f = t_f = r_f = d_f = k_f = 0;
  polite_f = polite = 1; 
*/
  
  /* "Production" opts to be read in from command line or ini file */
  int q = -1;
  int device_number = -1;
  checkpoint_iter = -1;
  threads = -1;
  fftlen = -1;
  s_f = t_f = d_f = k_f = -1;
  polite_f = polite = -1;
  input_filename[0] = RESULTSFILE[0] = 0; /* First character is null terminator */
  
  /* Non-"production" opts */
  r_f = 0;
  int cufftbench_s, cufftbench_e, cufftbench_d;  
  cufftbench_s = cufftbench_e = cufftbench_d = 0;

  parse_args(&argc, &argv, &q, &device_number, &cufftbench_s, &cufftbench_e, &cufftbench_d);
  /* The rest of the args are globals */
  
  if (file_exists(INIFILE))
  {  
   if( checkpoint_iter < 1 && 		!IniGetInt(INIFILE, "CheckpointIterations", &checkpoint_iter, 10000) )
    fprintf(stderr, "Warning: Couldn't parse ini file option CheckpointIterations; using default: 10000\n");
   if( threads < 1 && 			!IniGetInt(INIFILE, "Threads", &threads, 256) )
    fprintf(stderr, "Warning: Couldn't parse ini file option Threads; using default: 256\n");
   if( s_f < 0 && 			!IniGetInt(INIFILE, "SaveAllCheckpoints", &s_f, 0) )
    fprintf(stderr, "Warning: Couldn't parse ini file option SaveAllCheckpoints; using default: off\n");
   if( 		     	     s_f > 0 && !IniGetStr(INIFILE, "SaveFolder", folder, "savefiles") )
    fprintf(stderr, "Warning: Couldn't parse ini file option SaveFolder; using default: \"savefiles\"\n");
   if( t_f < 0 && 			!IniGetInt(INIFILE, "CheckRoundoffAllIterations", &t_f, 0) )
    fprintf(stderr, "Warning: Couldn't parse ini file option CheckRoundoffAllIterations; using default: off\n");
   if( polite < 0 && 			!IniGetInt(INIFILE, "Polite", &polite, 1) )
    fprintf(stderr, "Warning: Couldn't parse ini file option Polite; using default: 1\n");
   if( k_f < 0 && 			!IniGetInt(INIFILE, "Interactive", &k_f, 0) )
    fprintf(stderr, "Warning: Couldn't parse ini file option Interactive; using default: off\n");
   if( device_number < 0 &&		!IniGetInt(INIFILE, "DeviceNumber", &device_number, 0) )
    fprintf(stderr, "Warning: Couldn't parse ini file option DeviceNumber; using default: 0\n");
   if( d_f < 0 &&			!IniGetInt(INIFILE, "PrintDeviceInfo", &d_f, 0) )
    fprintf(stderr, "Warning: Couldn't parse ini file option PrintDeviceInfo; using default: off\n");
   if( !input_filename[0] &&		!IniGetStr(INIFILE, "WorkFile", input_filename, "worktodo.txt") )
    fprintf(stderr, "Warning: Couldn't parse ini file option WorkFile; using default \"worktodo.txt\"\n");
    /* I've readded the warnings about worktodo and results due to the planned multiple-instances-in-one-dir feature. */
   if( !RESULTSFILE[0] && 		!IniGetStr(INIFILE, "ResultsFile", RESULTSFILE, "results.txt") )
    fprintf(stderr, "Warning: Couldn't parse ini file option ResultsFile; using default \"results.txt\"\n");
   if( fftlen < 0 && 			!IniGetInt(INIFILE, "FFTLength", &fftlen, 0) )
    fprintf(stderr, "Warning: Couldn't parse ini file option FFTLength; using autoselect.\n");
  }
  else // no ini file
    {
      fprintf(stderr, "Warning: Couldn't find .ini file. Using defaults for non-specified options.\n");
      if( checkpoint_iter < 1 ) checkpoint_iter = 10000;
      if( threads < 1 ) threads = 256;
      if( fftlen < 0 ) fftlen = 0;
      if( s_f < 0 ) s_f = 0;
      if( t_f < 0 ) t_f = 0;
      if( k_f < 0 ) k_f = 0;
      if( device_number < 0 ) device_number = 0;
      if( d_f < 0 ) d_f = 0;
      if( polite < 0 ) polite = 1;
      if( !input_filename[0] ) sprintf(input_filename, "worktodo.txt");
      if( !RESULTSFILE[0] ) sprintf(RESULTSFILE, "result.txt");
  }
  
  if (polite == 0) {
    polite_f = 0;
    polite = 1;
  } else {
    polite_f = 1;
  }
  if (threads != 32 && threads != 64 && threads != 128
	      && threads != 256 && threads != 512 && threads != 1024)
  {
    fprintf(stderr, "Error: thread count is invalid.\n");
    fprintf(stderr, "Threads must be 2^k, 5 <= k <= 10.\n");
    exit(2);
  }
  
  init_device (device_number);

  if (r_f)
    {
      fftlen = 0;
      checkpoint_iter = 10000;
      t_f = 1;
      check (86243, "23992ccd735a03d9");
      check (132049, "4c52a92b54635f9e");
      check (216091, "30247786758b8792");
      check (756839, "5d2cbe7cb24a109a");
      check (859433, "3c4ad525c2d0aed0");
      check (1257787, "3f45bf9bea7213ea");
      check (1398269, "a4a6d2f0e34629db");
      check (2976221, "2a7111b7f70fea2f");
      check (3021377, "6387a70a85d46baf");
      check (6972593, "88f1d2640adb89e1");
      check (13466917, "9fdc1f4092b15d69");
      check (20996011, "5fc58920a821da11");
      check (24036583, "cbdef38a0bdc4f00");
      check (25964951, "62eb3ff0a5f6237c");
      check (30402457, "0b8600ef47e69d27");
      check (32582657, "02751b7fcec76bb1");
      check (37156667, "67ad7646a1fad514");
      check (42643801, "8f90d78d5007bba7");
      check (43112609, "e86891ebf6cd70c4");
      if (bad_selftest)
      {
        fprintf(stderr, "Error: There ");
        bad_selftest > 1 ? fprintf(stderr, "were %d bad selftests!\n",bad_selftest) 
        		 : fprintf(stderr, "was a bad selftest!\n");
      }
    }
  else if (cufftbench_d)
    cufftbench (cufftbench_s, cufftbench_e, cufftbench_d);
  else
    {
      if (s_f)
	{
#ifdef linux
	  mode_t mode = S_IRWXU | S_IRGRP | S_IXGRP | S_IROTH | S_IXOTH;
	  if (mkdir (folder, mode) != 0)
	    fprintf (stderr,
		     "mkdir: cannot create directory `%s': File exists\n",
		     folder);
#else
	  if (_mkdir (folder) != 0)
	    fprintf (stderr,
		     "mkdir: cannot create directory `%s': File exists\n",
		     folder);
#endif
	}
      if (q <= 0)
      {
        int error;
	LINE_BUFFER AID; //! Assignment key; not useful as of yet
	#ifdef EBUG
	printf("Processed INI file and console arguments correctly; about to call get_next_assignment().\n");
	#endif
	do { //! while(!quitting)
  	  error = get_next_assignment(input_filename, &q, &AID, 1); //! Use default verbosity of 1
	  if( error ) exit (2); 
	  //! get_next_assignment prints warning message
	  #ifdef EBUG
	  printf("Gotten assignment, about to call check(). (This is really weird if you're seeing this.)\n");
	  #endif
	  check (q, 0);
	  
	  if(!quitting) //! Only clear assignment if not killed by user, i.e. test finished 
	    {
	      error = clear_assignment(input_filename, q);
	      if(error) {
	        if( error==3 )
	          fprintf(stderr, "Can't open workfile %s\n\n", input_filename);
	        else if( error==4 )
	          fprintf(stderr, "Can't open tmp workfile\n\n");
	        else if( error==5 )
	          fprintf(stderr, "Assignment M%d completed but not found in workfile\n\n", q);
	        else if( error==6 )
	          fprintf(stderr, "Cannot move tmp workfile to regular workfile\n\n");
	        exit (2);
	      } //! No error
	    } //! Not quitting
	  } while(!quitting);  
    } else //! Exponent passed in as argument
	{
	  if (!valid_assignment(q)) {printf("\n");} //! v_a prints warning
	  else {
	    check (q, 0);
	  }
	}
    }
}

void parse_args(int* _argc, char* *_argv[], int* q, int* device_number, 
		int* cufftbench_s, int* cufftbench_e, int* cufftbench_d)
{
int argc = *_argc;
char** argv = *_argv; /* Dereference the pointers */

while (argc > 1)
    {
      if (strcmp (argv[1], "-t") == 0)
	{
	  t_f = 1;
	  argv++;
	  argc--;
	}
      else if (strcmp (argv[1], "-h") == 0)
        {
      	  fprintf (stderr,
	       "$ CUDALucas -h|-v\n");
      	  fprintf (stderr,
	       "$ CUDALucas [-d device_number] [-info] [-i inifile] [-threads 32|64|128|256|512|1024] [-c checkpoint_iteration] [-f fft_length] [-s folder] [-t] [-polite iteration] [-k] exponent|input_filename\n");
      	  fprintf (stderr,
	       "$ CUDALucas [-d device_number] [-info] [-i inifile] [-threads 32|64|128|256|512|1024] [-t] [-polite iteration] -r\n");
      	  fprintf (stderr,
	       "$ CUDALucas [-d device_number] [-info] -cufftbench start end distance\n");
	  fprintf (stderr,
	       "                       -h print this help message\n");
	  fprintf (stderr,
	       "                       -info print device information\n");
	  fprintf (stderr,
	       "                       -i set .ini file name (default = \"CUDALucas.ini\")\n");
      	  fprintf (stderr,
	       "                       -threads set threads number (default=256)\n");
      	  fprintf (stderr,
	       "                       -f set fft length (if round off error then exit)\n");
      	  fprintf (stderr,
	       "                       -s save all checkpoint files\n");
      	  fprintf (stderr,
	       "                       -t check round off error all iterations\n");
      	  fprintf (stderr,
	       "                       -polite GPU polite per iteration (default -polite 1) -polite 0 GPU aggressive\n");
      	  fprintf (stderr,
	       "                       -cufftbench exec CUFFT benchmark (Ex. $ ./CUDALucas -d 1 -cufftbench 1179648 6291456 32768 )\n");
      	  fprintf (stderr, 
      	       "                       -r exec residue test.\n");
      	  fprintf (stderr,
	       "                       -k enable keys (p change -polite, t disable -t, s change -s)\n\n");
      	  exit (2);          
      	}
      else if (strcmp (argv[1], "-v") == 0)
        {  
          printf("%s\n\n", program);
          exit (2);
        }
      else if (strcmp (argv[1], "-polite") == 0)
	{
	  if (argc < 3)
	    {
	      fprintf (stderr, "can't parse -polite option\n\n");
	      exit (2);
	    }
	  polite = atoi (argv[2]);
	  if (polite == 0)
	    {
	      polite_f = 0;
	      polite = 1;
	    }
	  argv += 2;
	  argc -= 2;
	}
      else if (strcmp (argv[1], "-r") == 0)
	{
	  r_f = 1;
	  argv++;
	  argc--;
	}
      else if (strcmp (argv[1], "-k") == 0)
	{
	  k_f = 1;
	  argv++;
	  argc--;
	}
      else if (strcmp (argv[1], "-d") == 0)
	{
	  if (argc < 3)
	    {
	      fprintf (stderr, "can't parse -d option\n\n");
	      exit (2);
	    }
	  *device_number = atoi (argv[2]);
	  argv += 2;
	  argc -= 2;
	}
      else if (strcmp (argv[1], "-i") == 0)
	{
	  if(argc < 3)
	    {
	      fprintf (stderr, "can't parse -i option\n\n");
	      exit (2);
	    }
	  sprintf (INIFILE, "%s", argv[2]);
	  argv += 2;
	  argc -= 2;
	}
      else if (strcmp (argv[1], "-info") == 0)
        {
          d_f = 1;
          argv++;
          argc--;
        }
      else if (strcmp (argv[1], "-cufftbench") == 0)
	{
	  if (argc < 5)
	    {
	      fprintf (stderr, "can't parse -cufftbench option\n\n");
	      exit (2);
	    }
	  *cufftbench_s = atoi (argv[2]);
	  *cufftbench_e = atoi (argv[3]);
	  *cufftbench_d = atoi (argv[4]);
	  argv += 4;
	  argc -= 4;
	}
      else if (strcmp (argv[1], "-threads") == 0)
	{
	  if (argc < 3)
	    {
	      fprintf (stderr, "can't parse -threads option\n\n");
	      exit (2);
	    }
	  threads = atoi (argv[2]);
	  if (threads != 32 && threads != 64 && threads != 128
	      && threads != 256 && threads != 512 && threads != 1024)
	    {
	      fprintf(stderr, "Error: thread count is invalid.\n");
	      fprintf(stderr, "Threads must be 2^k, 5 <= k <= 10.\n\n");
	      exit (2);
	    }
	  argv += 2;
	  argc -= 2;
	}
      else if (strcmp (argv[1], "-c") == 0)
	{
	  if (argc < 3)
	    {
	      fprintf (stderr, "can't parse -c option\n\n");
	      exit (2);
	    }
	  checkpoint_iter = atoi (argv[2]);
	  if (checkpoint_iter == 0)
	    {
	      fprintf (stderr, "can't parse -c option\n\n");
	      exit (2);
	    }
	  argv += 2;
	  argc -= 2;
	}
      else if (strcmp (argv[1], "-f") == 0)
	{
	  if (argc < 3)
	    {
	      fprintf (stderr, "can't parse -f option\n\n");
	      exit (2);
	    }
	  fftlen = atoi (argv[2]);
	  argv += 2;
	  argc -= 2;
	}
      else if (strcmp (argv[1], "-s") == 0)
	{
	  s_f = 1;
	  if (argc < 3)
	    {
	      fprintf (stderr, "can't parse -s option\n\n");
	      exit (2);
	    }
	  sprintf (folder, "%s", argv[2]);
	  argv += 2;
	  argc -= 2;
	}
      else
	{
	  if (*q != -1 || strcmp (input_filename, "") != 0 )
	    {
	      fprintf (stderr, "can't parse options\n\n");
	      exit (2);
	    }
	  int derp = atoi (argv[1]);
	  if (derp == 0) {
	    sprintf (input_filename, "%s", argv[1]);
	  } else { *q = derp; }
	  argv++;
	  argc--;
	}
    }
}
